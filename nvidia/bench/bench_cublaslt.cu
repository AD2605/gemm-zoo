#include "kernel_functors/cublaslt_gemm.cuh"
#include "utils/benchmark.cuh"

#include <hip/hip_runtime.h>

#include <tuple>

int main() {
  float alpha = 1.0f;
  float beta = 1.0f;

  using kernel_functor_struct =
      nvidia::kernel_functors::cublasLt_gemm<float, float>;
  benchmark::benchmark<kernel_functor_struct, float, float>(
      {{1024, 1024, 1024}, {2048, 2048, 2048}, {4096, 4096, 4096}}, alpha,
      beta);
}
